
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/resource.h>

#define TILE_SIZE 16
#define SIZE (TILE_SIZE * 256)
#define MALLOC_MATRIX(n) (float*)malloc((n)*(n)*sizeof(float))

float* device_malloc(int n){
	float* m;
	if(hipMalloc(&m, n*n*sizeof(float)) == hipErrorOutOfMemory) return NULL;
	return m;
}

__global__ void gpuPower(float* res, float* inp, int n, int power){
	__shared__ float s_res[TILE_SIZE*TILE_SIZE];
	__shared__ float s_inp[TILE_SIZE*TILE_SIZE];
	__shared__ float s_tmp[TILE_SIZE*TILE_SIZE];
	const unsigned int row = blockIdx.y*blockDim.y + threadIdx.y;
	const unsigned int col = blockIdx.x*blockDim.x + threadIdx.x;
	int pow = 0, idx = 0;
	float sum = 0.0f;

	if(row < n && col < n){
		//copy the input to its share
		s_inp[row*n+col] = inp[row*n+col];
		//set the shared result to the identity matrix
		if(row == col) s_res[row*n+col] = 1.0f;
		else s_res[row*n+col]=0.0f;
	}
	__syncthreads();

	for(pow=0; pow<power; pow++){
		//multiply the matrices
		if(row < n && col < n){
			sum = 0.0f;
			for(idx=0; idx<n; idx++){
				sum = sum + s_res[row*n+idx]*s_inp[idx*n+col];
			}
			s_tmp[row*n+col] = sum;
		}
		__syncthreads();
		//copy temp back to result
		if(row < n && col < n) s_res[row*n+col] = s_tmp[row*n+col];
		__syncthreads();
	}
	__syncthreads();

	//copy the result matrix to global res
	if(row < n && col < n) res[row*n+col] = s_res[row*n+col];
	__syncthreads();
}

void power_gpu(float* result, float* input, int n, int power){
	dim3 bdim(TILE_SIZE, TILE_SIZE);
	dim3 gdim(SIZE/TILE_SIZE, SIZE/TILE_SIZE);

	float* d_result = device_malloc(n);
	float* d_input = device_malloc(n);

	hipMemcpy(d_input, input, n*n*sizeof(float), hipMemcpyHostToDevice);

	gpuPower<<<gdim, bdim>>>(d_result, d_input, n, power);

	hipMemcpy(result, d_result, n*n*sizeof(float), hipMemcpyDeviceToHost);

	hipFree(d_result);
	hipFree(d_input);
}

void power_cpu(float* result, float* input, int n, int power){
	int pow;
	int i, k, r, c;
	float* temp = MALLOC_MATRIX(n);

	//set result to be the identity matrix
	for(i=0; i<n*n; i++){
		if(i/n == i%n) result[i] = 1.0f;
		else result[i] = 0.0f;
	}

	for(pow=0; pow<power; pow++){
		//multiply result*input -> temp
		for(r=0; r<n; r++) for(c=0; c<n; c++){
			float sum = 0;
			for(k=0; k<n; k++){
				sum = sum + result[r*n+k]*input[k*n+c];
			}
			temp[r*n+c] = sum;
		}
		//copy temp back to result
		for(i=0; i<n*n; i++){
			result[i] = temp[i];
		}
	}
}

void printMat(float* mat, int n){
	for(int r=0; r<n; r++){
		for(int c=0; c<n; c++){
			if(c != 0) printf(" ");
			printf("%.3f", mat[r*n+c]);
		}
		printf("\n");
	}
}

int main() {
	int n;
	int power;
	float* mat = NULL;
	float* result = NULL;

	scanf("%d", &n);
	scanf("%d", &power);

	if(power < 0 || n < 0){
		printf("error\n");
		return 1;
	}

	mat = MALLOC_MATRIX(n);
	result = MALLOC_MATRIX(n);
	if(!mat || !result){
		printf("error\n");
		return 1;
	}

	for(int i=0; i<(n*n); i++){
		if(!scanf("%f", &(mat[i]))){
			printf("error\n");
			free(mat);
			free(result);
			return 1;
		}
	}

	power_gpu(result, mat, n, power);

	printMat(mat, n);
	printMat(result, n);
	free(mat);
	free(result);
	return 0;
}

